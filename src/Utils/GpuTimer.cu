#include "Utils/GpuTimer.cuh"
#include "Assert.cuh"


float Gpu::Timer::timeMs;
hipEvent_t Gpu::Timer::startEvent;
hipEvent_t Gpu::Timer::stopEvent;


void Gpu::Timer::initialize() {
  cuCheck(hipEventCreate(&startEvent));
  cuCheck(hipEventCreate(&stopEvent));
  timeMs = 0;
}

void Gpu::Timer::reset() {
  timeMs = 0;
}
    
float Gpu::Timer::getTimeMs() {
  return timeMs;
}
    
void Gpu::Timer::start(hipEvent_t& event, hipStream_t stream) {
  cuCheck(hipEventRecord(event, stream));
}

float Gpu::Timer::stop(hipEvent_t& s_event, hipEvent_t& e_event, hipStream_t stream) {
  float ms;
  cuCheck(hipEventRecord(e_event, stream));
  cuCheck(hipEventSynchronize(e_event));
  cuCheck(hipEventElapsedTime(&ms, s_event, e_event));
  timeMs += ms;
  return ms;
}
